#include "hip/hip_runtime.h"
// Matrix Multiplication in CUDA

#include <stdio.h>
//#include <string.h>
//#include <assert.h>
//#include <stdlib.h>
#include <hip/hip_runtime.h>


// includes, project
////////////////////////////////////////////////////////////////////////////////
// declarations, forward

#define WIDTH 32 
extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

// FILL HERE: define constant variable


// MatrixMul kernel
/**
 * CUDA Kernel Device code
 *
 * Computes the matrix multiplication of A and B into C. The 3 matrices have the same
 * number of elements WIDTH*WIDTH.
 */
// FILL HERE: translate C-version matrixMul to CUDA-version kernel code
void 
MatrixMul(float* A, float* B, float* C)
{

    // TODO : Kernel Function
    //        C = A * B
    // --> 
    for(int i = 0; i < WIDTH; i++)
	{
		for(int j = 0; j < WIDTH; j++)
		{
			for(int k = 0; k < WIDTH; k++)
    		{
        		C[i*WIDTH + j] += A[i*WIDTH+k] * B[k*WIDTH + j];
    		}
		}
	}
    // <--    

}

/**
 * Host main routine
 */
int 
main(void) 
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the matrix size to be used, and compute its size
    int size = WIDTH*WIDTH*sizeof(float);
    printf("[MatrixMul of %d x %d elements]\n", WIDTH, WIDTH);

    // Allocate the host input matrix h_A 
    float  *h_A = (float *)malloc(size);

    // Allocate the host input matrix h_B 
    float  *h_B = (float *)malloc(size);

    // Allocate the host input matrix h_C 
    float  *h_C = (float *)malloc(size);

    // Allocate the host matrix for compute check 
    float  *reference = (float *)malloc(size);
    
    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL || reference == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrices!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input matrices
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < WIDTH; ++j)
        {
            h_A[i*WIDTH + j] = 0.01f;
            h_B[i*WIDTH + j] = 1.0f;
        }
    }
    memset(h_C, 0, size);
    memset(reference, 0, size);

    // compute the matrix multiplication on the CPU for comparison
    computeGold(reference, h_A, h_B, WIDTH, WIDTH, WIDTH);

	// Allocate device input matrices 
	// TODO : Leave/Remove the given hipMalloc code properly
	// --> 
	float* d_A = NULL;	
    err = hipMalloc((void**)&d_A, size); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	float* d_B = NULL;	
    err = hipMalloc((void**)&d_B, size); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	// <--

	// Allocate the device output matrix
	float* d_C = NULL;
    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input matrix A and B in host memory to the device input matrices in
    // device memory
	// TODO : Add proper mem copy APIs according to the memory that matrix A and B will be stored
	// -->
    printf("Copy input data from the host memory to the CUDA device\n");
    err = ;// FILL HERE
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = ;// FILL HERE
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	// <--

	// TODO : Clock Measurements
	//		  Add code to return clock cycles from kernel
	// -->
#ifdef TM
    unsigned long long* d_runtime;
    int r_size = WIDTH*WIDTH*sizeof(unsigned long long);
    unsigned long long* runtime = (unsigned long long*)malloc(r_size);
    memset(runtime, 0, r_size);
    hipMalloc((void**)&d_runtime, r_size);
#endif
	// <--

    // TODO : Kernel Invocation 
    //        Assign as many threads as the size of matrix in a thread block and
    //        invoke the kernel function.
    // --> 
    int blocksPerGrid = ;// FILL HERE
    int threadsPerBlock = ;// FILL HERE
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	MatrixMul(d_A, d_B, d_C);
    // <--                                                           

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matrixMul kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	hipDeviceSynchronize();
	
    // Copy the device result matrix in device memory to the host result matrix
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    // Verify that the result matrix is correct
    bool res = 1;
	for (int i = 0; i < WIDTH*WIDTH; i++)
	{
		float diff = fabs(reference[i] - h_C[i]);
		if(diff > 0.001f)
		{
			res = 0;
			break;
		}
	}
	printf("Test %s\n", (res == 1) ? "PASSED" : "FAILED");

	// TODO : Get elapsed clock cycles from device to host
	//		  Take the longest time as kernel execution time
	// -->
#ifdef TM
    hipMemcpy(runtime, d_runtime, r_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    unsigned long long elapsed_time = 0;
    for(int i = 0; i < WIDTH*WIDTH; i++)
        if(elapsed_time < runtime[i])
            elapsed_time = runtime[i];
    printf("Kernel Execution Time: %llu cycles\n", elapsed_time);
#endif
	// <--

	// TODO : Free device global memory
	// 		  Leave/Remove the given hipFree statement according to your data allocation
	// -->
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
#ifdef TM
    hipFree(d_runtime);
#endif
	// <--

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(reference);
#ifdef TM
	free(runtime);
#endif

	return 0;
}

void
computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }

}
